#include "hip/hip_runtime.h"
___global___ void add(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(void){
	int a, b, c;		// host copies
	int *dA, *dB, *dC; 	// device copies
	int size = sizeof(int);
	
	// Allocate space for device copies
	hipMalloc((void **) &dA, size);
	hipMalloc((void **) &dB, size);
	hipMalloc((void **) &dC, size);
	
	// setup input values
	a =2;
	b =7;
	
	// copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	
	// Launch add()
	add<<<1,1>>(dA, dB, dC);
	
	// Copy back to host
	hipMemcpy(&c, dC, size, hipMemcpyDeviceToHost);
	
	// cleanup
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	
	return 0;
	}