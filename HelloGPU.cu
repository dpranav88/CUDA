#include "hip/hip_runtime.h"
// nvcc HelloGPU.cu
// runs on gpu
___global___ void mykernel(void){  // global indicates runs on device and initiated by host
}

int main(void)
{
	mykernel<<<1,1>>>(); // <<<blks no,thrds no>>>
						 // <<< makes call from host code to device code, also called kernel launch
	Print("Hello World!\n");
	return 0;
}
