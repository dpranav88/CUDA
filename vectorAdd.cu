#include "hip/hip_runtime.h"
___global___ void add(int *a, int *b, int *c){
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512

int main(void){
	int a, b, c;		// host copies
	int *dA, *dB, *dC; 	// device copies
	int size = N * sizeof(int);
	
	// Allocate space for device copies
	hipMalloc((void **) &dA, size);
	hipMalloc((void **) &dB, size);
	hipMalloc((void **) &dC, size);
	
	// setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size); 
	
	// copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	
	// Launch add()
	add<<<N,1>>(dA, dB, dC);
	
	// Copy back to host
	hipMemcpy(&c, dC, size, hipMemcpyDeviceToHost);
	
	// cleanup
	free(a);
	free(b);
	free(c);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	
	return 0;
}